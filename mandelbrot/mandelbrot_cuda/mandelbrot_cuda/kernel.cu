#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <thrust/complex.h>

#include <stdio.h>
#include <iostream>

#include <SFML/Graphics.hpp>



struct set_area {
    double startX;
    double endX;
    double startY;
    double endY;
};

__device__
int mandelbrot_point(thrust::complex<double> c, int max_iterations)
{
    thrust::complex<double> z = thrust::complex<double>(0, 0);
    for (int i = 0; i < max_iterations; i++)
    {
        z = z * z + c;
        
        if (thrust::abs(z) > 4)
        {
            return i;
        }
    }
    return max_iterations;
}


__global__
void mandelbrot_set(int* m_set, set_area area, int num_points, int max_iterations)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int length = num_points * num_points;

    double spacing_x = abs(area.endX - area.startX) / ((double)num_points - 1.0);
    double spacing_y = abs(area.endY - area.startY) / ((double)num_points - 1.0);

    for (int i = index; i < length; i += stride)
    {
        int pos_x = i % num_points;
        int pos_y = i / num_points;
        double x = area.startX + pos_x * spacing_x;
        double y = area.startY + pos_y * spacing_y;

        thrust::complex<double> c = thrust::complex<double>(x, y);

        int iterations = mandelbrot_point(c, max_iterations);

        m_set[i] = iterations;
    }
}


int* point_color(int iterations, int min, int max)
{
    double percentage = (1.0 * (iterations - min)) / (max - min);
    int* colors = new int[4];
    colors[3] = 255;
    if (percentage <= 0.33)
    {
        percentage = percentage / 0.33;
        colors[0] = std::ceil(255.0 * percentage);
        colors[1] = 0;
        colors[1] = 0;
    }
    else if (percentage <= 0.66)
    {
        percentage = (0.66 - percentage) / 0.33;
        colors[0] = std::ceil(255.0 * percentage);
        colors[1] = std::ceil(255.0 * (1.0 - percentage));
        colors[2] = 0;
    }
    else
    {
        percentage = (1.0 - percentage) / 0.34;
        colors[0] = 0;
        colors[1] = std::ceil(255.0 * percentage);
        colors[2] = std::ceil(255.0 * (1.0 - percentage));
    }

    return colors;
}



sf::Uint8* set_to_image(int* m_set, int num_points, int max_iterations)
{
    int min = max_iterations;
    int max = 0;
    for (int x = 0; x < num_points * num_points; x++)
    {
        min = std::min(min, m_set[x]);
        max = std::max(max, m_set[x]);
    }

    sf::Uint8* image = new sf::Uint8[(long long)num_points * num_points * 4];
    for (int x = 0; x < num_points * num_points; x++)
    {
        int index = x * 4;
        int* color = point_color(m_set[x], min, max);
        image[index] = color[0];
        image[index + 1] = color[1];
        image[index + 2] = color[2];
        image[index + 3] = color[3];
    }
    return image;
}


set_area refresh_mandelbrot(int* m_set, int num_points, int max_iterations, set_area previousArea, int x, int y, int zoomDelta, int numBlocks, int blockSize)
{
    double factor = 1;
    double rangeX = previousArea.endX - previousArea.startX;
    double rangeY = previousArea.endY - previousArea.startY;
    double factorX = (double) x / num_points - 0.5;
    double factorY = (double) y / num_points - 0.5;

    previousArea.startX += factorX * rangeX;
    previousArea.endX += factorX * rangeX;
    previousArea.startY += factorY * rangeY;
    previousArea.endY += factorY * rangeY;

    double centerX = previousArea.startX + rangeX / 2.0;
    double centerY = previousArea.startY + rangeY / 2.0;

    if (zoomDelta > 0)
    {
        factor = 1.0 / (zoomDelta + 1);
    }
    else if (zoomDelta < 0) {
        factor = std::abs(zoomDelta - 1);
    }
    previousArea.startX = centerX - (rangeX / 2.0) * factor;
    previousArea.endX = centerX + (rangeX / 2.0) * factor;
    previousArea.startY = centerY - (rangeY / 2.0) * factor;
    previousArea.endY = centerY + (rangeY / 2.0) * factor;

    mandelbrot_set<<<numBlocks, blockSize>>>(m_set, previousArea, num_points, max_iterations);
    hipDeviceSynchronize();

    return previousArea;
}

int main()
{
    constexpr int num_points = 1000;
    constexpr int base_iterations = 50;
    size_t size_old = num_points * num_points;
    size_t size = num_points * num_points * sizeof(int);

    set_area area;
    area.startX = -2.25;
    area.endX = 0.75;
    area.startY = -1.5;
    area.endY = 1.5;

    int blockSize = 256;
    int numBlocks = (num_points + blockSize - 1) / blockSize;
    int* m_set;
    hipMallocManaged(&m_set, size);
    mandelbrot_set << <numBlocks, blockSize >> > (m_set, area, num_points, base_iterations);
    hipDeviceSynchronize();

    constexpr int width = num_points;
    constexpr int height = num_points;
    sf::RenderWindow window(sf::VideoMode(width, height), "It works!");
    
    sf::Uint8* image = set_to_image(m_set, num_points, 120);


    sf::Texture texture;
    if (!texture.create(width, height))
    {
        return -1;
    }
    texture.update(image);
    sf::Sprite sprite;
    sprite.setTexture(texture);
    int total_zoom = 0;
    while (window.isOpen())
    {
        sf::Event event;
        while (window.pollEvent(event))
        {
            if (event.type == sf::Event::Closed)
                window.close();

            if (event.type == sf::Event::MouseWheelScrolled)
            {
                if (event.mouseWheelScroll.wheel == sf::Mouse::VerticalWheel)
                {
                    int x = event.mouseWheelScroll.x;
                    int y = event.mouseWheelScroll.y;
                    int zoom = event.mouseWheelScroll.delta;
                    total_zoom += zoom;
                    int max_iterations = std::max(base_iterations, base_iterations * total_zoom);
                    std::cout << "Iter: " << max_iterations << std::endl;
                    area = refresh_mandelbrot(m_set, num_points, max_iterations, area, x, y, zoom, numBlocks, blockSize);
                    image = set_to_image(m_set, num_points, max_iterations);
                    texture.update(image);
                }
            }
        }
        window.clear();
        window.draw(sprite);
        window.display();
    }

    hipFree(m_set);
    return 0;
}