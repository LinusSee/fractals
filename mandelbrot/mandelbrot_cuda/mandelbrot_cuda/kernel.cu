#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <complex>
#include <tuple>

#include <SFML/Graphics.hpp>

using namespace std;


int mandelbrot_point(complex<double> c, int max_iterations)
{
    std::complex<double> z(0, 0);
    for (int i = 0; i < max_iterations; i++)
    {
        z = z * z + c;
        if (abs(z) > 4)
        {
            return i;
        }
    }
    return max_iterations;
}

int** mandelbrot_set(double start_x, double end_x, double start_y, double end_y, int num_points, int max_iterations)
{
    double spacing_x = abs(end_x - start_x) / (num_points - 1);
    double spacing_y = abs(end_y - start_y) / (num_points - 1);

    cout << "SpacingX: " << spacing_x << endl;
    cout << "SpacingY: " << spacing_y << endl;

    double current_x = start_x;
    double current_y = start_y;

    int** m_set = new int* [num_points];
    for (int i = 0; i < num_points; i++)
    {
        m_set[i] = new int[num_points];
        for (int j = 0; j < num_points; j++)
        {

            complex<double> c(current_x, current_y);
            int iterations = mandelbrot_point(c, max_iterations);

            m_set[i][j] = iterations;
            current_x = current_x + spacing_x;
        }
        current_x = start_x;
        current_y = current_y + spacing_y;
    }

    return m_set;
}

int* point_color(int iterations, int max_iterations)
{
    float percentage = (1.0f * iterations) / max_iterations;
    int* colors = new int[4];
    colors[3] = 255;
    if (percentage <= 0.33f)
    {
        percentage = percentage / 0.33f;
        colors[0] = std::ceil(255 * percentage);
        colors[1] = 0;
        colors[1] = 0;
    }
    else if (percentage <= 0.66f)
    {
        percentage = (0.66f - percentage) / 0.33f;
        colors[0] = std::ceil(255 * percentage);
        colors[1] = std::ceil(255 * (1 - percentage));
        colors[2] = 0;
    }
    else
    {
        percentage = (1.0f - percentage) / 0.34f;
        colors[0] = 0;
        colors[1] = std::ceil(255 * percentage);
        colors[2] = std::ceil(255 * (1 - percentage));
    }

    return colors;
}







sf::Uint8* set_to_image(int** m_set, int num_points, int max_iterations)
{
    sf::Uint8* image = new sf::Uint8[num_points * num_points * 4];
    for (int x = 0; x < num_points; x++)
    {
        for (int y = 0; y < num_points; y++)
        {
            int index = x * num_points * 4 + y * 4;
            //sf::Uint8* color = point_color(m_set[x][y], max_iterations);
            int* color = point_color(m_set[x][y], max_iterations);
            image[index] = color[0];
            image[index + 1] = color[1];
            image[index + 2] = color[2];
            image[index + 3] = color[3];
        }
    }
    return image;
}


int main()
{
    constexpr int num_points = 1000;
    int** m_set = mandelbrot_set(-2.25, 0.75, -1.5, 1.5, num_points, 120);


    constexpr float width = num_points;
    constexpr float height = num_points;
    sf::RenderWindow window(sf::VideoMode(num_points, num_points), "It works!");


    sf::Uint8* image = set_to_image(m_set, num_points, 120);

    sf::Texture texture;
    if (!texture.create(num_points, num_points))
    {
        return -1;
    }
    texture.update(image);
    sf::Sprite sprite;
    sprite.setTexture(texture);

    while (window.isOpen())
    {
        sf::Event event;
        while (window.pollEvent(event))
        {
            if (event.type == sf::Event::Closed)
                window.close();
        }

        window.clear();
        window.draw(sprite);
        window.display();
    }
    return 0;
}
